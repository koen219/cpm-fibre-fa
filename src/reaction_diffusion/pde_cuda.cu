#include "hip/hip_runtime.h"
#include "pde.cpp"
#include "pde_cuda.cuh"
// Allow the CPU implementation for the PDE solver when CUDA is enabled.
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#define ARRAY_SIZE 1
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }



__device__ void DerivativesPDE(PDEFIELD_TYPE current_time, PDEFIELD_TYPE *y,
                               PDEFIELD_TYPE *dydt, int *sigmafield, int id,
                               PDEFIELD_TYPE *secr_rate,
                               PDEFIELD_TYPE *decay_rate) {
  int sigma = sigmafield[id];
  if (sigma > 0) {
    dydt[0] = secr_rate[0];
  } else {
    // outside cells
    dydt[0] = -decay_rate[0] * y[0];
  }
}


void cuErrorChecker(hipError_t errSync, hipError_t errAsync) {
  errSync = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}



void PDE::InitialiseCuda() {
  cout << "Start cuda init" << endl;

  hipMalloc((void **)&d_diffusioncoefficient,
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE));
  hipMalloc((void **)&d_celltype, sizex * sizey * sizeof(int));
  hipMalloc((void **)&d_sigmafield, sizex * sizey * sizeof(int));

  hipMalloc((void **)&d_PDEvars,
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_PDEvars, PDEvars, layers * sizex * sizey * sizeof(PDEFIELD_TYPE),
             hipMemcpyHostToDevice);
  hipMalloc((void **)&d_alt_PDEvars,
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_alt_PDEvars, alt_PDEvars,
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE),
             hipMemcpyHostToDevice);
  hipMalloc((void **)&d_secr_rate, ARRAY_SIZE * sizeof(PDEFIELD_TYPE));
  hipMalloc((void **)&d_decay_rate, ARRAY_SIZE * sizeof(PDEFIELD_TYPE));
  #ifdef PDEFIELD_DOUBLE
  hipMemcpy(d_secr_rate, par.secr_rate.data(),
             ARRAY_SIZE * sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMemcpy(d_decay_rate, par.decay_rate.data(),
             ARRAY_SIZE * sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  #else
  vector<float> float_secr_rate (par.secr_rate.begin(), par.secr_rate.end());
  vector<float> float_decay_rate (par.decay_rate.begin(), par.decay_rate.end());
  hipMemcpy(d_secr_rate, float_secr_rate.data(),
             ARRAY_SIZE * sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMemcpy(d_decay_rate, float_decay_rate.data(),
             ARRAY_SIZE * sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);

  #endif


  // Needed for ADI steps
  gpuErrchk(hipMallocManaged(&upperH, sizex * sizey * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagH, sizex * sizey * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerH, sizex * sizey * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BH, sizex * sizey * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&upperV, sizey * sizex * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagV, sizey * sizex * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerV, sizey * sizex * sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BV, sizey * sizex * sizeof(PDEFIELD_TYPE)));

  handleH = 0;
  pbuffersizeH = 0;
  pbufferH = NULL;
  statusH = hipsparseCreate(&handleH);
#ifdef PDEFIELD_DOUBLE
  hipsparseDgtsvInterleavedBatch_bufferSizeExt(
      handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey,
      &pbuffersizeH); // Compute required buffersize for horizontal sweep
#else
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(
      handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey,
      &pbuffersizeH); // Compute required buffersize for horizontal sweep
#endif
  gpuErrchk(hipMalloc(&pbufferH, sizeof(char) * pbuffersizeH));

  handleV = 0;
  pbuffersizeV = 0;
  pbufferV = NULL;
  statusV = hipsparseCreate(&handleV);
#ifdef PDEFIELD_DOUBLE
  hipsparseDgtsvInterleavedBatch_bufferSizeExt(
      handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex,
      &pbuffersizeV); // Compute required buffersize for vertical sweep
#else
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(
      handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex,
      &pbuffersizeV); // Compute required buffersize for vertical sweep
#endif
  gpuErrchk(hipMalloc(&pbufferV, sizeof(char) * pbuffersizeV));

  cout << "End cuda init" << endl;
}


__global__ void InitialiseDiagonals(int sizex, int sizey,
                                    PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2,
                                    PDEFIELD_TYPE *lowerH,
                                    PDEFIELD_TYPE *upperH, PDEFIELD_TYPE *diagH,
                                    PDEFIELD_TYPE *lowerV,
                                    PDEFIELD_TYPE *upperV, PDEFIELD_TYPE *diagV,
                                    PDEFIELD_TYPE *diffusioncoefficient) {
  // This function could in theory be parellelized further, split into 6 (each
  // part only assigning 1 value.), but this is probably slower
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc; // position we currently want to assign to
  int yloc;
  int idcc; // id corresponding to the diffusioncoefficient (+sizey to get the
            // value right, +1 to get the value above)
  for (int id = index; id < sizex * sizey; id += stride) {
    xloc = id / sizey; // needed to obtain interleaved format
    yloc = id % sizey;
    idcc = xloc * sizey + yloc;
    if (xloc == 0) {
      lowerH[id] = 0;
      diagH[id] = diffusioncoefficient[idcc + sizey] / dx2 + twooverdt;
      upperH[id] = -diffusioncoefficient[idcc + sizey] / dx2;
    } else if (xloc == sizex - 1) {
      lowerH[id] = -diffusioncoefficient[idcc - sizey] / dx2;
      diagH[id] = diffusioncoefficient[idcc - sizey] / dx2 + twooverdt;
      upperH[id] = 0;
    } else {
      lowerH[id] = -diffusioncoefficient[idcc - sizey] / dx2;
      diagH[id] = (diffusioncoefficient[idcc + sizey] +
                   diffusioncoefficient[idcc - sizey]) /
                      dx2 +
                  twooverdt;
      upperH[id] = -diffusioncoefficient[idcc + sizey] / dx2;
    }

    xloc = id % sizex; // needed to obtain interleaved format
    yloc = id / sizex;
    idcc = xloc * sizey + yloc;
    if (yloc == 0) {
      lowerV[id] = 0;
      diagV[id] = diffusioncoefficient[idcc + 1] / dx2 + twooverdt;
      upperV[id] = -diffusioncoefficient[idcc + 1] / dx2;
    } else if (yloc == sizey - 1) {
      lowerV[id] = -diffusioncoefficient[idcc - 1] / dx2;
      diagV[id] = diffusioncoefficient[idcc - 1] / dx2 + twooverdt;
      upperV[id] = 0;
    } else {
      lowerV[id] = -diffusioncoefficient[idcc - 1] / dx2;
      diagV[id] =
          (diffusioncoefficient[idcc + 1] + diffusioncoefficient[idcc - 1]) /
              dx2 +
          twooverdt;
      upperV[id] = -diffusioncoefficient[idcc + 1] / dx2;
    }
  }
}

__global__ void InitialiseHorizontalVectors(int sizex, int sizey,
                                            PDEFIELD_TYPE twooverdt,
                                            PDEFIELD_TYPE dx2,
                                            PDEFIELD_TYPE *BH,
                                            PDEFIELD_TYPE *diffusioncoefficient,
                                            PDEFIELD_TYPE *alt_PDEvars) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; // id corresponding to the diffusioncoefficient and
            // alt_PDEvars(+sizey to get the value right, +1 to get the value
            // above)
  for (int id = index; id < sizex * sizey; id += stride) {
    xloc = id / sizey; // needed to obtain interleaved format
    yloc = id % sizey;
    idcc = xloc * sizey + yloc;

    if (yloc == 0)
      BH[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc + 1] *
                (alt_PDEvars[idcc + 1] - alt_PDEvars[idcc])) /
                   dx2;
    else if (yloc == sizey - 1)
      BH[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc - 1] *
                (alt_PDEvars[idcc - 1] - alt_PDEvars[idcc])) /
                   dx2;
    else
      BH[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc + 1] *
                    (alt_PDEvars[idcc + 1] - alt_PDEvars[idcc]) +
                diffusioncoefficient[idcc - 1] *
                    (alt_PDEvars[idcc - 1] - alt_PDEvars[idcc])) /
                   dx2;
  }
}


__global__ void InitialiseVerticalVectors(int sizex, int sizey,
                                          PDEFIELD_TYPE twooverdt,
                                          PDEFIELD_TYPE dx2, PDEFIELD_TYPE *BV,
                                          PDEFIELD_TYPE *diffusioncoefficient,
                                          PDEFIELD_TYPE *alt_PDEvars) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; // id corresponding to the diffusioncoefficient and
            // alt_PDEvars(+sizey to get the value right, +1 to get the value
            // above)
  for (int id = index; id < sizex * sizey; id += stride) {
    xloc = id % sizex;
    yloc = id / sizex;
    idcc = xloc * sizey + yloc;

    if (xloc == 0)
      BV[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc + sizey] *
                (alt_PDEvars[idcc + sizey] - alt_PDEvars[idcc])) /
                   dx2;
    else if (xloc == sizex - 1)
      BV[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc - sizey] *
                (alt_PDEvars[idcc - sizey] - alt_PDEvars[idcc])) /
                   dx2;
    else
      BV[id] = twooverdt * alt_PDEvars[idcc] +
               (diffusioncoefficient[idcc + sizey] *
                    (alt_PDEvars[idcc + sizey] - alt_PDEvars[idcc]) +
                diffusioncoefficient[idcc - sizey] *
                    (alt_PDEvars[idcc - sizey] - alt_PDEvars[idcc])) /
                   dx2;
  }
}

__global__ void NewPDEfieldH0(
    int sizex, int sizey, PDEFIELD_TYPE *BH,
    PDEFIELD_TYPE *PDEvars) { // Take the values from BH and assign the new
                              // values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex * sizey; id += stride) {
    PDEvars[id] = BH[id];
  }
}

__global__ void NewPDEfieldV0(
    int sizex, int sizey, PDEFIELD_TYPE *BV,
    PDEFIELD_TYPE *PDEvars) { // Take the values from BV and assign the new
                              // values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex * sizey; id += stride) {
    PDEvars[sizey * (id % sizex) + id / sizex] =
        BV[id]; // Conversion is needed because PDEvars iterates over columns
                // first and then rows, while BV does the opposite
  }
}

__global__ void NewPDEfieldOthers(
    int sizex, int sizey, int layers, PDEFIELD_TYPE *BV, PDEFIELD_TYPE *PDEvars,
    PDEFIELD_TYPE
        *alt_PDEvars) { // copy the other values from alt_PDEvars to PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = 0; id < layers * sizex * sizey; id += stride) {
    PDEvars[id] = alt_PDEvars[id];
  }
}

__global__ void ODEstepFE(PDEFIELD_TYPE dt, PDEFIELD_TYPE ddt, double thetime,
                          int layers, int sizex, int sizey,
                          PDEFIELD_TYPE *PDEvars, PDEFIELD_TYPE *alt_PDEvars,
                          int *sigmafield, PDEFIELD_TYPE *secr_rate,
                          PDEFIELD_TYPE *decay_rate) {

  int nr_of_iterations = round(dt / ddt);
  if (fabs(dt / ddt - nr_of_iterations) > 0.001)
    printf("dt and ddt do not divide properly!");
  if (nr_of_iterations == 0)
    printf("0 iterations!");
  PDEFIELD_TYPE stepsize;
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE current_time;
  PDEFIELD_TYPE MaxTimeError = 5e-7;
  int i;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex * sizey; id += stride) {
    current_time = thetime;
    for (i = 0; i < layers; i++)
      y[i] = PDEvars[i * sizex * sizey + id];
    for (int it = 0; it < nr_of_iterations; it++) {
      DerivativesPDE(current_time, y, dydt, sigmafield, id, secr_rate,
                     decay_rate);
      current_time += ddt;
      if (it == nr_of_iterations - 1) { // Are we done?
        for (i = 0; i < layers; i++) {
          alt_PDEvars[i * sizex * sizey + id] = y[i] + ddt * dydt[i];
        }
      } else {
        for (i = 0; i < layers; i++) {
          y[i] = y[i] + ddt * dydt[i];
        }
      }
    }
  }
}

__global__ void CopyAltToOriginalPDEvars(int sizex, int sizey, int layers,
                                         PDEFIELD_TYPE *PDEsource,
                                         PDEFIELD_TYPE *PDEtarget) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers * sizex * sizey; id += stride) {
    PDEtarget[id] = PDEsource[id];
  }
}

void PDE::cuPDEsteps(CellularPotts *cpm, int repeat) {
  // copy current diffusioncoefficient matrix and celltype matrix from host to
  // device
  hipError_t errSync;
  hipError_t errAsync;
  sigmafield = cpm->getSigma();
  hipMemcpy(d_diffusioncoefficient, DiffCoeffs[0][0],
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE),
             hipMemcpyHostToDevice);
  hipMemcpy(d_sigmafield, sigmafield[0], sizex * sizey * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_PDEvars, PDEvars[0][0],
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE),
             hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  errSync = hipGetLastError();
  errAsync = hipDeviceSynchronize();

  int nr_blocks = sizex * sizey / par.threads_per_core + 1;

  for (int iteration = 0; iteration < repeat / repeat; iteration++) {

    // setup matrices for upperdiagonal, diagonal and lower diagonal for both
    // the horizontal and vertical direction, since these remain the same during
    // one PDE step
    InitialiseDiagonals<<<par.number_of_cores, par.threads_per_core>>>(
        sizex, sizey, 2 / dt, dx2, lowerH, upperH, diagH, lowerV, upperV, diagV,
        d_diffusioncoefficient);
    hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    cuODEstep();
    cuHorizontalADIstep();
    thetime = thetime + dt / 2;
    cuODEstep();
    cuVerticalADIstep();
    thetime = thetime + dt / 2;
  }
  hipMemcpy(PDEvars[0][0], d_PDEvars,
             layers * sizex * sizey * sizeof(PDEFIELD_TYPE),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

void PDE::cuODEstep() {
  // Do an ODE step of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  ODEstepFE<<<par.number_of_cores, par.threads_per_core>>>(
      dt / 2, ddt, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars,
      d_sigmafield, d_secr_rate, d_decay_rate);
  cuErrorChecker(errSync, errAsync);
}

void PDE::cuHorizontalADIstep() {
  // Do a horizontal ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitialiseHorizontalVectors<<<par.number_of_cores, par.threads_per_core>>>(
      sizex, sizey, 2 / dt, dx2, BH, d_diffusioncoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
#ifdef PDEFIELD_DOUBLE
  statusH = hipsparseDgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH,
                                          upperH, BH, sizey, pbufferH);
#else
  // statusH = hipsparseSgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH,
  // upperH, BH, sizey, pbufferH);
#endif
  if (statusH != HIPSPARSE_STATUS_SUCCESS) {
    cout << statusH << endl;
  }
  NewPDEfieldH0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BH,
                                                               d_PDEvars);
  cuErrorChecker(errSync, errAsync);
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(
      sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);
}

void PDE::cuVerticalADIstep() {
  // Do a vertical ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitialiseVerticalVectors<<<par.number_of_cores, par.threads_per_core>>>(
      sizex, sizey, 2 / dt, dx2, BV, d_diffusioncoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
#ifdef PDEFIELD_DOUBLE
  statusV = hipsparseDgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV,
                                          upperV, BV, sizex, pbufferV);
#else
  statusV = hipsparseSgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV,
                                          upperV, BV, sizex, pbufferV);
#endif
  if (statusV != HIPSPARSE_STATUS_SUCCESS) {
    cout << statusV << endl;
  }
  hipDeviceSynchronize();
  NewPDEfieldV0<<<par.number_of_cores, par.threads_per_core>>>(
      sizex, sizey, BV, d_PDEvars); //////
  errSync = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(
      sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);
}
